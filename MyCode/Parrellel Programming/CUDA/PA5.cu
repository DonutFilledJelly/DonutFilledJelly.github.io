#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 1024
#define T 1024
//simple variance calculation.

//This esssentially adds all values together by first
//adding 1 and 2 to 1, 3 and 4 to 3,
//then after the first loop, adding 1 and 3 to 1,
//meaning by the end, 1 is the sum of all the array
__global__ void sum(float *a, int count){
    
    int scope;//scope is for checking to see what level of addition we are on
    int blocknum = blockDim.x * blockIdx.x;//find block id
    int threadnum = threadIdx.x;
    for(scope = 1; scope <= T; scope*=2){//Each level of addition
        int threadmax = T/scope;
        if(threadnum < threadmax){
            int first = blocknum*2 + threadnum*scope*2;
            int second = first + scope;
            if(first < count && second < count){
                a[first] += a[second];//adding
            }
        }
        __syncthreads();
        
    }
}

__global__ void sumagain(float *a, int count){
    
    int scope;//scope is for checking to see what level of addition we are on
    int blocknum = blockDim.x * blockIdx.x;//find block id
    int threadnum = threadIdx.x;
    for(scope = 1; scope <= T; scope*=2){//Each level of addition
        int threadmax = T/scope;
        if(threadnum < threadmax){
            int first = blocknum*2 + threadnum*scope*2;
            int second = first + scope;
            if(first < count && second < count){
                a[first] += a[second];//adding
            }
        }
        __syncthreads();
        
    }
}
//adds the last few together bettween blocks.
__global__ void finishUp(float *a, int count){
    for (int i = T*2; i < count; i += T*2)//has to be T*2
    {
        a[0] += a[i];
    }
}

//Varaince equation (Excluding division)
__global__ void variancething(float *a, int count, float average){
    int where = blockDim.x * blockIdx.x + threadIdx.x;
    if(where < count){// avoid segfault
        a[where] = pow(a[where]-average, 2);
    }
    __syncthreads();
}



int main(){
    float *a;//float array for host
	float *d_a;//float array for device
	int count = 0;//amount in the array
	scanf("%d", &count);
	int size = count * sizeof(float);//size of arrays
    
    
    a = (float *)malloc(size); 
	hipMalloc(&d_a, size);
	
    for(int i = 0; i < count; i++){//reading in array
        scanf("%f", &a[i]);
    }
    
    
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);//sending array to gpu
    sum<<<N, T>>>(d_a, count);//add all into d_a[0]
    finishUp<<<1,1>>>(d_a, count);//add between blocks
    
    float sum; 
    hipMemcpy(&sum, d_a, sizeof(float), hipMemcpyDeviceToHost);
    float average = sum/count;//average is the total sum over the amount of numbers
    printf("The average is %f\n", average);
    
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    variancething<<<N, T>>>(d_a, count, average);//finding variance in next three lines
    sumagain<<<N, T>>>(d_a, count);//need to create a new sum, not sure why but syntax wont let me. Exactly same as last sum
    finishUp<<<1, 1>>>(d_a, count);
    
    float variance = 0;
    hipMemcpy(&variance, d_a, sizeof(float), hipMemcpyDeviceToHost);
    variance = variance/(count-1);//finding last bit of variance (division)
    
    
    printf("Variance is %f\n", variance);
    
	
    free(a); 
    hipFree(d_a);

    return 0;
}
